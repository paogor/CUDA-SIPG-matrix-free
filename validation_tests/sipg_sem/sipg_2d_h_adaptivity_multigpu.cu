#include "hip/hip_runtime.h"
/***
* /file 
*
* Validation test for Laplacian matrix.
* 
*   solve  - u" = f  with Dirichlet boundary conditions
*   Numerical solution is confronted with analytical one.
*
*/

#include<iostream>
#include<fstream>
#include<sstream>
#include<string>
#include<vector>

#include<mpi.h>

#define USE_PRECONDITIONER 
#define USE_MODE_MATRIX
#define MPI_NODE_PER_EDGE 32 

#define EXACT_SOLUTION_NO 5
#include"../analytical_solutions.hpp"

#include<sipg_sem_2d_multigpu.hpp>

#include<iomanip>

#include<CUDA_TIMER.hpp>

int main(int argc, char** argv)
{


  MPI_Init(&argc, &argv);

  int pid, nprocs;
  MPI_Comm CartComm;
  MPI_Comm_rank(MPI_COMM_WORLD, &pid);
  MPI_Comm_size(MPI_COMM_WORLD, &nprocs);

  const double toll = 1e-7;


  if(pid == 0)
  {
    std::cerr<<"EXACT_SOLUTION_NO "<<EXACT_SOLUTION_NO<<std::endl;
    std::cerr<<"MPI_NODE_PER_EDGE "<<MPI_NODE_PER_EDGE<<std::endl;
#ifdef USE_MODE_MATRIX
    std::cout<<"USE_MODE_MATRIX is ON"<<std::endl;
#endif
#ifdef USE_PRECONDITIONER
    std::cout<<"USE_PRECONDITIONER is ON"<<std::endl;
#endif
    std::cout<<"toll: "<<toll<<std::endl;
  }

  int dims[3] = {MPI_NODE_PER_EDGE, MPI_NODE_PER_EDGE, 1};
  int period[3] = {0, 0, 0};

  MPI_Cart_create(MPI_COMM_WORLD, 3, dims, period, false, &CartComm);

  int coords[3] = {0, 0, 0};
  MPI_Cart_get(CartComm, 3, dims, period, coords);

  int degree = 2;
//  for (int degree = 3; degree < 5; ++degree)
  {

    double L2_err_old(0), H1_err_old(0);

//  int dim = 8;
    for (int dim = 32; dim < 2049; dim*=2)
    {

      CUDA_TIMER t;
      using namespace test_func;
      square_mesh_multigpu<double> sq_mesh( dim, MPI_NODE_PER_EDGE, coords[0], coords[1] ); 

      if (pid == 0) t.start();
      sipg_sem_2d_multigpu<double> p(CartComm, degree, sq_mesh, f, u_ex, dx_u_ex, dy_u_ex, toll);
      if (pid == 0) t.stop();


      if(pid == 0)
      {

        std::cerr<<MPI_NODE_PER_EDGE*dim<<"\t"<<degree<<"\t";
        std::cerr<<std::setw(12)<<log(p.H1_err/H1_err_old)/log(2)<<"\t";
        std::cerr<<std::setw(12)<<p.H1_err<<"\t";
        std::cerr<<std::setw(12)<<log(p.L2_err/L2_err_old)/log(2)<<"\t";
        std::cerr<<std::setw(12)<<p.L2_err<<"\t";
        std::cerr<<t.elapsed_millisecs();
        std::cerr<<"\t"<<p.iterations;
        std::cerr<<std::endl;

        L2_err_old = p.L2_err;
        H1_err_old = p.H1_err;

      }
 
      sq_mesh.device_info.free();

   }

  if (pid == 0) std::cerr<<std::endl;

}



#if 0
  hipError_t error = hipGetLastError();
  std::string lastError = hipGetErrorString(error); 
  std::cout<<lastError<<std::endl;
#endif

  MPI_Finalize();

  return 0;

}


